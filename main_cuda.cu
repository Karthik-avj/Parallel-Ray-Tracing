
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Image Dimensions
#define N 1920
#define M 1080

// Finite Light Dimensions
#define XL_MIN 4.5
#define XL_MAX 5.5
#define ZL_MIN 4.5
#define ZL_MAX 5.5
#define YL 5.0

// Shadow rays to Light
#define L_RANDOM 16

// Number of reflections
#define MAX_DEPTH 3

// Code constants
#define NUM_OBJ 4
#define OBJ_LEN 15

// Global functions
#define MAX(x,y) ( ((x) > (y)) ? x : y )
#define MIN(x,y) ( ((x) < (y)) ? x : y )


// finding the ray direction for a given origin and destination
__device__ void ray_direction(float* origin, float* point, 
                              float* vector){
    float dr[3];
    dr[0] = point[0]-origin[0];
    dr[1] = point[1]-origin[1];
    dr[2] = point[2]-origin[2];

    float norm = sqrt(dr[0]*dr[0] + dr[1]*dr[1] + dr[2]*dr[2]);
    vector[0] = dr[0]/norm;
    vector[1] = dr[1]/norm;
    vector[2] = dr[2]/norm;
}


// finding reflected ray for given normal and ray direction
__device__ void reflected_direction(float* incoming,
                    float* normal, float* reflected){
    float dr[3];
    float dp = incoming[0]*normal[0] + incoming[1]*normal[1]
               + incoming[2]*normal[2];
    dr[0] = incoming[0] - 2*dp*normal[0];
    dr[1] = incoming[1] - 2*dp*normal[1];
    dr[2] = incoming[2] - 2*dp*normal[2];

    float norm = sqrt(dr[0]*dr[0] + dr[1]*dr[1] + dr[2]*dr[2]);
    reflected[0] = dr[0]/norm;
    reflected[1] = dr[1]/norm;
    reflected[2] = dr[2]/norm;
}


// distance between origin and intersection of sphere if any
__device__ void sphere_intersection(float* origin, 
                    float* ray_direction, float* center, 
                    float* radius, float* dist){
    float b, c, disc;
    float dr[3];
    dr[0] = origin[0]-center[0];
    dr[1] = origin[1]-center[1];
    dr[2] = origin[2]-center[2];

    b = 2 * (ray_direction[0]*dr[0]+
    ray_direction[1]*dr[1]+
    ray_direction[2]*dr[2]);

    c = dr[0]*dr[0] + dr[1]*dr[1] + dr[2]*dr[2] - *radius * *radius;

    disc = b*b - (4*c);
    if(disc<=0){
        *dist = -1.0;
        return;
    }
    else{
        float val1, val2;
        float sqd = sqrt(disc);
        val1 = (-b + sqd)/2;
        val2 = (-b - sqd)/2;
        if(val1>0 && val2>0){
            *dist = fmin(val1, val2);
            return;
        }
        else{
            *dist = -1.0;
            return;
        }
    }
}

// Nearest object which intersects with light ray
__device__ void nearest_intersection_object(float *objects, 
                    float *origin, float *ray_direction, 
                    float *min_dist,  int *object_idx){
	float distances[NUM_OBJ];
	for(int i=0; i<(NUM_OBJ); i++){
		float center[] = {objects[i*OBJ_LEN+0], 
                          objects[i*OBJ_LEN+1],
                          objects[i*OBJ_LEN+2]};
		sphere_intersection(origin, ray_direction, center, 
                            &objects[i*OBJ_LEN+3], &distances[i]);
	}

	for(int i=0; i<(NUM_OBJ); i++){
		if((distances[i]<(*min_dist)) && (distances[i]>0)){
			*min_dist = distances[i];
			*object_idx = i;
		}
	}
}

// Checks if the point is in shadow of other objects
__device__ void shadowed(int *is_shad, float *normal, float *light_dir, 
              float *shifted_point, float *min_dist, float *origin, 
              float *ray_dir, float *light_source, float *objects, 
              int *object_idx){

    float intersection_point[3];
    for (int i=0; i<3; i++){
        intersection_point[i] = *min_dist * ray_dir[i] + origin[i];
    }

    float object_center[] = {objects[*object_idx * OBJ_LEN],
                             objects[*object_idx * OBJ_LEN + 1],
                             objects[*object_idx * OBJ_LEN + 2]};
    ray_direction(object_center, intersection_point, normal);


    for (int i=0; i<3; i++){
        shifted_point[i] = 0.0001 * normal[i] + intersection_point[i];
    }

    ray_direction(shifted_point, light_source, light_dir);

    float min_distance = __INT_MAX__;
    int useless = -1;
    nearest_intersection_object(objects, shifted_point, light_dir,
                                &min_distance, &useless);

    float light_dist[3];
    light_dist[0] = light_source[0]-intersection_point[0];
    light_dist[1] = light_source[1]-intersection_point[1];
    light_dist[2] = light_source[2]-intersection_point[2];
    float intersection_to_light_dist = sqrt(
        (light_dist[0])*(light_dist[0]) + 
        (light_dist[1])*(light_dist[1]) + 
        (light_dist[2])*(light_dist[2]));

    if (min_distance < intersection_to_light_dist) {
        *is_shad = 1;
    }
    else{
        *is_shad = 0;
    }
}

// Determines the color using Blinn-Phong reflection model
__device__ void color(float* normal_surface, float* light_intersection,
                    float* ray_dir, float* object, float* light, 
                    float* reflection, float* illumination){

    float ambient[3] = {object[4]*light[5],
                        object[5]*light[6],
                        object[6]*light[7]};

    float nl_dp = normal_surface[0] * light_intersection[0] +
                  normal_surface[1] * light_intersection[1] +
                  normal_surface[2] * light_intersection[2];

    float diffuse[3] = {object[7]*light[5]*nl_dp,
                        object[8]*light[6]*nl_dp,
                        object[9]*light[7]*nl_dp};

    float light_ray[3] = {light_intersection[0]-ray_dir[0],
                          light_intersection[1]-ray_dir[1],
                          light_intersection[2]-ray_dir[2]};

    float norm = sqrt(
        light_ray[0]*light_ray[0] +
        light_ray[1]*light_ray[1] +
        light_ray[2]*light_ray[2]);

    float nlr_dp = normal_surface[0] * light_ray[0] +
                   normal_surface[1] * light_ray[1] +
                   normal_surface[2] * light_ray[2];

    nlr_dp = nlr_dp / norm;
    nlr_dp = pow(nlr_dp, 0.25*object[13]);

    float specular[3] = {object[10]*light[8]*nlr_dp,
                         object[11]*light[9]*nlr_dp,
                         object[12]*light[10]*nlr_dp};

    illumination[0] += *reflection *(ambient[0] + diffuse[0] + specular[0]);
    illumination[1] += *reflection *(ambient[1] + diffuse[1] + specular[1]);
    illumination[2] += *reflection *(ambient[2] + diffuse[2] + specular[2]);
}


// calculation of a single pixel (main gpu kernel)
__global__ void single_pixel(float* objects, float* lights, 
                             float* camera, float* screen, int* image,
                             int N_full, int M_full){
    // indices using blockIdx and threadIdx
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int col = blockIdx.y*blockDim.y+threadIdx.y;

    // sanity check for unnecessary calculation
    if((row > N_full) || (col > M_full))
        return;
    
    float ray_dir[3];
    float origin[3];
    float dx = (screen[1] - screen[0]) / N_full;
    float dy = (screen[3] - screen[2]) / M_full;
    float illumination[3] = {};

    float point[] = {screen[0] + dx * row, screen[2] + dy * col, 0};
    float single_object[OBJ_LEN];

    origin[0] = camera[0];
    origin[1] = camera[1];
    origin[2] = camera[2];

    ray_direction(origin, point, ray_dir);
    float reflection = 1.0;

    for (int k=0; k < MAX_DEPTH; k++){
        float min_dist = __INT_MAX__;
        int n_object_idx = -1;

        nearest_intersection_object(objects, origin, ray_dir, 
                                    &min_dist, &n_object_idx);

        if (n_object_idx == -1){
            break;
        }
        
        int is_shad;
        float normal[3], light_dir[3], shifted_point[3];
        
        for (int i=0; i<OBJ_LEN; i++){
            single_object[i] = objects[n_object_idx*OBJ_LEN + i];
        }

        hiprandState_t state;
        hiprand_init(M_full*row+N_full*col, 0, 0, &state);
        
        for (int l=0; l<L_RANDOM; l++){
            // generating random light position
            float x_rand = hiprand_uniform(&state);
            float z_rand = hiprand_uniform(&state);

            float light_pos[] = {
                lights[0] + x_rand*(lights[1] - lights[0]),
                lights[4],
                lights[2] + z_rand*(lights[3] - lights[2])};
            shadowed(&is_shad, normal, light_dir, shifted_point, 
                     &min_dist, origin, ray_dir, light_pos, objects, 
                     &n_object_idx);
            
            if (is_shad == 1)
                continue;
    
            color(normal, light_dir, ray_dir, single_object, 
                  lights, &reflection, illumination);
    
        }
        
        reflection *= single_object[14];
        origin[0] = shifted_point[0];
        origin[1] = shifted_point[1];
        origin[2] = shifted_point[2];

        reflected_direction(ray_dir, normal, ray_dir);
    }

    // storing the color in image (clipping between 0 and 255) (PPM format)
    image[3 * M_full * row + 3*col + 0] = MIN(MAX(0, sqrt(illumination[0]/(L_RANDOM))), 1)*255;
    image[3 * M_full * row + 3*col + 1] = MIN(MAX(0, sqrt(illumination[1]/(L_RANDOM))), 1)*255;
    image[3 * M_full * row + 3*col + 2] = MIN(MAX(0, sqrt(illumination[2]/(L_RANDOM))), 1)*255;
}


// MAIN Function
int main(){
    // objects being used in image
    float objects[] = {
        -0.2, 0, -1, 0.7, 0.1, 0, 0, 0.7, 0, 0, 1, 1, 1, 100, 0.5,
        0.1, -0.3, 0, 0.1, 0.1, 0, 0.1, 0.7, 0, 0.7, 1, 1, 1, 100, 0.5,
        -0.3, 0, 0, 0.15, 0, 0.1, 0, 0, 0.6, 0, 1, 1, 1, 100, 0.5,
        -0.2, -9000, -1, 8999.3, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 100, 0
        };
    
    // global variables
    float light[] = {XL_MIN, XL_MAX, ZL_MIN, ZL_MAX, YL, 1, 1, 1, 1, 1, 1, 1, 1, 1};
    float camera[] = {0, 0, 1};
    
    // 9 points in each pixel
    int N_full = 2*N+1;
    int M_full = 2*M+1;
    float screen[] = {-1.0, 1.0, -(float)M_full/N_full, (float)M_full/N_full};
    
    // image_final is the final image
    int *image, *image_final;
    int size_image_final = N*M*3*sizeof(int);
    int size_image = N_full*M_full*3*sizeof(int);
    image = (int*)malloc(size_image);
    image_final = (int*)malloc(size_image_final);

    int size_objects = NUM_OBJ*OBJ_LEN*sizeof(float);

    // device varibles
    float *dev_objects, *dev_light, *dev_camera, *dev_screen;
    int *dev_image;

    // allocating device variables
    hipMalloc((void**) &dev_objects, size_objects);
    hipMalloc((void**) &dev_light, 12*sizeof(float));
    hipMalloc((void**) &dev_camera, 3*sizeof(float));
    hipMalloc((void**) &dev_image, size_image);
    hipMalloc((void**) &dev_screen, 4*sizeof(float));

    // copying from host to device
    hipMemcpy(dev_objects, objects, size_objects, hipMemcpyHostToDevice);
    hipMemcpy(dev_light, light, 12*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_camera, camera, 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_screen, screen, 4*sizeof(float), hipMemcpyHostToDevice);

    // defining the dimensions of blocks and grids
    dim3 blockDim(16, 16);
    int k, l;
    if (N_full%blockDim.x == 0) {
        k = N_full/blockDim.x;
    } else {
        k = N_full/blockDim.x + 1;
    }
    if (M_full%blockDim.x == 0) {
        l = M_full/blockDim.x;
    } else {
        l = M_full/blockDim.x + 1;
    }
    dim3 gridDim(k, l);

    // kernel
    single_pixel<<<gridDim, blockDim>>>(dev_objects, dev_light, dev_camera, dev_screen, dev_image, N_full, M_full);

    // copying from device to host
    hipMemcpy(image, dev_image, size_image, hipMemcpyDeviceToHost);

    // averaging the results of 9 points in each pixel
    for (int i=1; i<N_full-1; i+=2){
        for (int j=1; j<M_full-1; j+=2){

            int sum_red = 0;
            int sum_green = 0;
            int sum_blue = 0;

            for (int k=-1; k<=1; k++){
                for (int l=-1; l<=1; l++){
                    sum_red += image[3*M_full*(i+k)+3*(j+l)+0];
                    sum_green += image[3*M_full*(i+k)+3*(j+l)+1];
                    sum_blue += image[3*M_full*(i+k)+3*(j+l)+2];
                }
            }

            image_final[3*M*(i-1)/2 + 3*(j-1)/2 + 0] = sum_red/9;
            image_final[3*M*(i-1)/2 + 3*(j-1)/2 + 1] = sum_green/9;
            image_final[3*M*(i-1)/2 + 3*(j-1)/2 + 2] = sum_blue/9;
            
            }
    }

    // printing in PPM format
    printf("P3\n");
    printf("%d %d\n", N, M);
    printf("255 \n");
    for(int j=M-1; j>=0; j--){
        for(int i=0; i<N; i++){
            printf("%d %d %d\n", image_final[3*M*i+3*j+0], image_final[3*M*i+3*j+1], image_final[3*M*i+3*j+2]);
        }
    }

    hipFree(dev_objects);
    hipFree(dev_light);
    hipFree(dev_camera);
    hipFree(dev_image);
    hipFree(dev_screen);
    free(image);
    free(image_final);

    return 0;
}